
#include <hip/hip_runtime.h>
#include "stdint.h"

typedef uint32_t u32;
typedef unsigned long long int u64;

typedef std::pair<u64, u64> KV64;

constexpr u64 U64_MAX = ((u64)-1);

constexpr u64 kEmpty = U64_MAX;
constexpr u64 vEmpty = U64_MAX;

__host__ __device__ u64 hash64(u64 cap, u64 k) {
  k ^= k >> 33;
  k *= 0xff51afd7ed558ccd;
  k ^= k >> 33;
  k *= 0xc4ceb9fe1a85ec53;
  k ^= k >> 33;
  return k & (cap - 1);
}

// quick modulo for power of 2
__host__ __device__ static inline u64 p2mod(u64 x, u64 p2) {
  return x & (p2 - 1);
}

extern "C" __global__ void k_get(KV64 *hashtable, u64 capacity, u64 *ins,
                                 u64 *outs, u32 n_kvs) {

  u32 threadid = blockIdx.x * blockDim.x + threadIdx.x;
  u32 stride = blockDim.x * gridDim.x;

  for (u32 i = threadid; i < n_kvs; i += stride) {
    u64 key = ins[i];
    u64 slot = hash64(capacity, key);
    slot = p2mod(slot, capacity);

    while (true) {
      KV64 kv = hashtable[slot];

      if (kv.first == kEmpty || kv.second == vEmpty) {
        outs[i] = vEmpty;
        break;
      }
      if (kv.first == key) {
        outs[i] = kv.second;
        break;
      }

      slot = p2mod(slot + 1, capacity);
    }
  }
}

extern "C" __global__ void k_setup(KV64 *hashtable, u64 capacity) {

  u32 threadid = blockIdx.x * blockDim.x + threadIdx.x;
  u32 stride = blockDim.x * gridDim.x;

  for (u32 i = threadid; i < capacity; i += stride) {
    hashtable[i].second = hashtable[i].first + 1;
  }
}

extern "C" __global__ void k_insert(KV64 *hashtable, u64 capacity, KV64 *kvs,
                                    u32 n_kvs) {

  u32 threadid = blockIdx.x * blockDim.x + threadIdx.x;
  u32 stride = blockDim.x * gridDim.x;

  for (u32 i = threadid; i < n_kvs; i += stride) {
    KV64 kv = kvs[i];
    u64 slot = hash64(capacity, kv.first);
    slot = p2mod(slot, capacity);

    while (true) {
      u64 prev = atomicCAS(&hashtable[slot].first, kEmpty, kv.first);

      if (prev == kEmpty || prev == kv.first) {
        hashtable[slot].second = kv.second;
        break;
      }

      slot = p2mod(slot + 1, capacity);
    }
  }
}